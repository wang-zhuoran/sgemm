
#include <hip/hip_runtime.h>
#include<cstdio>
#include <iostream>
#include <chrono>
#include <cstring>       // for memset

#define A(i, j) a[(i) * n + (j)]
#define B(i, j) b[(i) * n + (j)]
#define abs(x) ((x) < 0.0 ? -(x) : (x))
void random_matrix(int m, int n, float* a) {
    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
#if 1
            A(i, j) = 2.0 * (float)drand48() - 1.0;
#else   
            A(i, j) = (j - i) % 3;
#endif 
        }
    }
}

float compare_matrices(int m, int n, float *a, float *b, int a_cols, int b_cols) {
    int i, j;
    float max_diff = 0.0, diff;
    int printed = 0;
  
    for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++) {
        float a_val = a[i * a_cols + j];  // 正确按 stride 访问
        float b_val = b[i * b_cols + j];
        diff = abs(a_val - b_val);
        max_diff = (diff > max_diff ? diff : max_diff);
        if (0 == printed)
          if (max_diff > 0.5f || max_diff < -0.5f) {
            printf("\n error: i %d  j %d diff %f  got %f  expect %f ", i, j, max_diff, a_val, b_val);
            printed = 1;
          }
      }
    }
  
    return max_diff;
}

// A: [M][K]
// B: [K][N]
// C: [M][N]
void cpu_sgemm(float* A_ptr, float* B_ptr, float* C_ptr, const int M, const int N, const int K) {
    for(int m = 0; m < M; m++) {
        for(int n = 0; n < N; n++) {
            float temp = 0.0f;
            for(int k = 0; k < K; k++) {
                temp += A_ptr[m * K + k] * B_ptr[k * N + n];
            }
            C_ptr[m * N + n] = temp;
        }
    }
}

/*

   ← s tile loop (K dimension)
    ┌────────────────────────────┐
    │  load A_tile to smem       │
    │  load B_tile to smem       │
    │  __syncthreads()           │
    │                            │
    │  ← k loop over tile depth  │
    │   ┌─────────────────────┐  │
    │   │  load A col to reg  │  │
    │   │  load B row to reg  │  │
    │   │  temp += A × Bᵀ     │  │ ← outer product
    │   └─────────────────────┘  │
    │                            │
    │  __syncthreads()           │
    └────────────────────────────┘

→ Write back temp[2][2] to global C

*/

#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])

// A: [M][K]
// B: [K][N]
// C: [M][N]
template <unsigned int M_NUM_PER_BLOCK, 
        unsigned int N_NUM_PER_BLOCK, 
        unsigned int K_NUM_PER_BLOCK, 
        unsigned int NUM_PER_THREAD>
__global__ void cuda_gemm(float* A_ptr, float* B_ptr, float* C_ptr, const int M, const int N, const int K) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty * blockDim.x + tx; // 重排
    int ctx = tid % 16;
    int cty = tid / 16;
    float* A_ptr_start = A_ptr + blockIdx.y * M_NUM_PER_BLOCK * K;
    float* B_ptr_start = B_ptr + blockIdx.x * N_NUM_PER_BLOCK;

    __shared__ float a_shared[M_NUM_PER_BLOCK][K_NUM_PER_BLOCK];
    __shared__ float b_shared[K_NUM_PER_BLOCK][N_NUM_PER_BLOCK];

    constexpr int REG_NUM = 2;
    float a_reg[REG_NUM] = {0.f};
    float b_reg[REG_NUM] = {0.f};

    float temp[REG_NUM][REG_NUM] = {0.f}; // 这里是因为外积会生成一个REG_NUM * REG_NUM的矩阵，然后累加这个矩阵才能得到最后的结果

    for(int s = 0; s < K; s += K_NUM_PER_BLOCK) {
        FETCH_FLOAT4(a_shared[ty][tx * NUM_PER_THREAD]) = FETCH_FLOAT4(A_ptr_start[K * ty + s + tx * NUM_PER_THREAD]);
        // a_shared[ty][tx * NUM_PER_THREAD + 0] = A_ptr_start[K * ty + s + tx * NUM_PER_THREAD + 0];
        // a_shared[ty][tx * NUM_PER_THREAD + 1] = A_ptr_start[K * ty + s + tx * NUM_PER_THREAD + 1];
        // a_shared[ty][tx * NUM_PER_THREAD + 2] = A_ptr_start[K * ty + s + tx * NUM_PER_THREAD + 2];
        // a_shared[ty][tx * NUM_PER_THREAD + 3] = A_ptr_start[K * ty + s + tx * NUM_PER_THREAD + 3];
        FETCH_FLOAT4(b_shared[ty][tx * NUM_PER_THREAD]) = FETCH_FLOAT4(B_ptr_start[(ty + s) * N + tx * NUM_PER_THREAD]);
        // b_shared[ty][tx * NUM_PER_THREAD + 0] = B_ptr_start[(ty + s) * N + tx * NUM_PER_THREAD + 0];
        // b_shared[ty][tx * NUM_PER_THREAD + 1] = B_ptr_start[(ty + s) * N + tx * NUM_PER_THREAD + 1];
        // b_shared[ty][tx * NUM_PER_THREAD + 2] = B_ptr_start[(ty + s) * N + tx * NUM_PER_THREAD + 2];
        // b_shared[ty][tx * NUM_PER_THREAD + 3] = B_ptr_start[(ty + s) * N + tx * NUM_PER_THREAD + 3];
        __syncthreads();
    
        for(int k = 0; k < K_NUM_PER_BLOCK; k++) {
            a_reg[0] = a_shared[cty * 2][k];
            a_reg[1] = a_shared[cty * 2 + 1][k];
            b_reg[0] = b_shared[k][ctx * 2];
            b_reg[1] = b_shared[k][ctx * 2 + 1];
            for(int i = 0; i < REG_NUM; i++) {
                for(int j = 0; j < REG_NUM; j++) {
                    temp[i][j] += a_reg[i] * b_reg[j];
                }
            }
        }
        __syncthreads();
    }

    float* C_ptr_start = C_ptr + blockIdx.y * M_NUM_PER_BLOCK * N + blockIdx.x * N_NUM_PER_BLOCK; 
    for(int i = 0; i < REG_NUM; i++) {
        for(int j = 0; j < REG_NUM; j++) {
            C_ptr_start[(cty * 2+ i) * N + (ctx * 2 + j)] = temp[i][j];
        }
    }
}

int main(){
    printf("Hello, SGEMM!\n");

    int m = 512;
    int n = 512;
    int k = 512;

    const size_t mem_size_A = m * k * sizeof(float);
    const size_t mem_size_B = k * n * sizeof(float);
    const size_t mem_size_C = m * n * sizeof(float);

    float* matrix_A_host = (float*)malloc(mem_size_A);
    float* matrix_B_host = (float*)malloc(mem_size_B);
    float* matrix_C_host_gpu_calc = (float*)malloc(mem_size_C);
    float* matrix_C_host_cpu_calc = (float*)malloc(mem_size_C);
    memset(matrix_A_host, 0, mem_size_A);
    memset(matrix_B_host, 0, mem_size_B);

    random_matrix(m, k, matrix_A_host);
    random_matrix(k, n, matrix_B_host);



    float* matrix_A_device, *matrix_B_device, *matrix_C_device;

    hipMalloc((void**)&matrix_A_device, mem_size_A);
    hipMalloc((void**)&matrix_B_device, mem_size_B);
    hipMalloc((void**)&matrix_C_device, mem_size_C);

    hipMemcpy(matrix_A_device, matrix_A_host, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(matrix_B_device, matrix_B_host, mem_size_B, hipMemcpyHostToDevice);

    // cpu_sgemm(matrix_A_host, matrix_B_host, matrix_C_host_cpu_calc, m, n, k);
    auto cpu_start = std::chrono::high_resolution_clock::now();
    cpu_sgemm(matrix_A_host, matrix_B_host, matrix_C_host_cpu_calc, m, n, k);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_duration = cpu_end - cpu_start;
    printf("CPU SGEMM time: %.3f ms\n", cpu_duration.count());


    // std::cout << &matrix_C_host_cpu_calc << std::endl;

    constexpr int M_NUM_PER_BLOCK = 32;
    constexpr int N_NUM_PER_BLOCK = 32;
    constexpr int K_NUM_PER_BLOCK = 32;
    constexpr int NUM_PER_THREAD = 4; // 每个线程负责多少个数字
    dim3 block(8, 32); // 原本是16,16 这里因为横向一次性取4个float所以改为8,32 thread总数仍然为256
    dim3 grid(m / M_NUM_PER_BLOCK, n / N_NUM_PER_BLOCK);
    // cuda_gemm<BLOCK, BLOCK><<<grid, block>>>(matrix_A_device, matrix_B_device, matrix_C_device, m, n, k);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    cuda_gemm<M_NUM_PER_BLOCK, N_NUM_PER_BLOCK, K_NUM_PER_BLOCK, NUM_PER_THREAD><<<grid, block>>>(matrix_A_device, matrix_B_device, matrix_C_device, m, n, k);
    hipEventRecord(stop);

    hipMemcpy(matrix_C_host_gpu_calc, matrix_C_device, mem_size_C, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU SGEMM time: %.9f ms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(matrix_C_host_gpu_calc, matrix_C_device, mem_size_C, hipMemcpyDeviceToHost);

    float diff = compare_matrices(m, n, matrix_C_host_gpu_calc, matrix_C_host_cpu_calc, k, n);

    printf("error: %f \n", diff);

    free(matrix_A_host);
    free(matrix_B_host);
    free(matrix_C_host_cpu_calc);
    free(matrix_C_host_gpu_calc);

    hipFree(matrix_A_device);
    hipFree(matrix_B_device);
    hipFree(matrix_C_device);

    return 0;
}