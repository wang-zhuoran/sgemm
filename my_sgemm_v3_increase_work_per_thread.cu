
#include <hip/hip_runtime.h>
#include<cstdio>
#include <iostream>
#include <chrono>
#include <cstring>       // for memset

#define A(i, j) a[(i) * n + (j)]
#define B(i, j) b[(i) * n + (j)]
#define abs(x) ((x) < 0.0 ? -(x) : (x))
void random_matrix(int m, int n, float* a) {
    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
#if 1
            A(i, j) = 2.0 * (float)drand48() - 1.0;
#else   
            A(i, j) = (j - i) % 3;
#endif 
        }
    }
}

float compare_matrices(int m, int n, float *a, float *b, int a_cols, int b_cols) {
    int i, j;
    float max_diff = 0.0, diff;
    int printed = 0;
  
    for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++) {
        float a_val = a[i * a_cols + j];  // 正确按 stride 访问
        float b_val = b[i * b_cols + j];
        diff = abs(a_val - b_val);
        max_diff = (diff > max_diff ? diff : max_diff);
        if (0 == printed)
          if (max_diff > 0.5f || max_diff < -0.5f) {
            printf("\n error: i %d  j %d diff %f  got %f  expect %f ", i, j, max_diff, a_val, b_val);
            printed = 1;
          }
      }
    }
  
    return max_diff;
}

// A: [M][K]
// B: [K][N]
// C: [M][N]
void cpu_sgemm(float* A_ptr, float* B_ptr, float* C_ptr, const int M, const int N, const int K) {
    for(int m = 0; m < M; m++) {
        for(int n = 0; n < N; n++) {
            float temp = 0.0f;
            for(int k = 0; k < K; k++) {
                temp += A_ptr[m * K + k] * B_ptr[k * N + n];
            }
            C_ptr[m * N + n] = temp;
        }
    }
}
// A: [M][K]
// B: [K][N]
// C: [M][N]
template <unsigned int BLOCK_SIZE, unsigned int STRIDE>
__global__ void cuda_gemm(float* A_ptr, float* B_ptr, float* C_ptr, const int M, const int N, const int K) {
    // const int x = blockDim.x * blockIdx.x + threadIdx.x;
    // const int y = blockDim.y * blockIdx.y + threadIdx.y;
    constexpr int STEP = BLOCK_SIZE * STRIDE;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    // if (x >= N || y >= M) return; 
    float* A_ptr_start = A_ptr + STEP * blockIdx.y * K; 
    float* B_ptr_start = B_ptr + STEP * blockIdx.x;

    __shared__ float a_shared[STEP][STEP];
    __shared__ float b_shared[STEP][STEP];

    float temp[STRIDE][STRIDE] = {0.f}; // 每个线程私有STRIDE * STRIDE 个temp临时数据

    for(int s = 0; s < K; s += STEP) {
        for(int i = 0; i < STRIDE; i++) {
            for(int j = 0; j < STRIDE; j++) {
                a_shared[ty + i * BLOCK_SIZE][tx + j * BLOCK_SIZE] = A_ptr_start[(ty + i * BLOCK_SIZE) * K + tx + j * BLOCK_SIZE + s];
                b_shared[ty + i * BLOCK_SIZE][tx + j * BLOCK_SIZE] = B_ptr_start[(ty + i * BLOCK_SIZE + s) * N + tx + j * BLOCK_SIZE];
            }
        }
        
        __syncthreads();

        for(int i = 0; i < STRIDE; i++) {
            for(int j = 0; j < STRIDE; j++) {
                for(int k = 0; k < STEP; k++) { // tile的大小是STEP*STEP
                    temp[i][j] += a_shared[ty + i * BLOCK_SIZE][k] * b_shared[k][tx + j * BLOCK_SIZE];
                }
            }
        }
        __syncthreads();
    }
    float* C_ptr_start = C_ptr + blockIdx.y * STEP * N + blockIdx.x * STEP; 
    for(int i = 0; i < STRIDE; i++) {
        for(int j = 0; j < STRIDE; j++) {
            C_ptr_start[(ty + i * BLOCK_SIZE) * N + tx + j * BLOCK_SIZE] = temp[i][j];
        }
    }
    
    
}

int main(){
    printf("Hello, SGEMM!\n");

    int m = 512;
    int n = 512;
    int k = 512;

    const size_t mem_size_A = m * k * sizeof(float);
    const size_t mem_size_B = k * n * sizeof(float);
    const size_t mem_size_C = m * n * sizeof(float);

    float* matrix_A_host = (float*)malloc(mem_size_A);
    float* matrix_B_host = (float*)malloc(mem_size_B);
    float* matrix_C_host_gpu_calc = (float*)malloc(mem_size_C);
    float* matrix_C_host_cpu_calc = (float*)malloc(mem_size_C);
    memset(matrix_A_host, 0, mem_size_A);
    memset(matrix_B_host, 0, mem_size_B);

    random_matrix(m, k, matrix_A_host);
    random_matrix(k, n, matrix_B_host);



    float* matrix_A_device, *matrix_B_device, *matrix_C_device;

    hipMalloc((void**)&matrix_A_device, mem_size_A);
    hipMalloc((void**)&matrix_B_device, mem_size_B);
    hipMalloc((void**)&matrix_C_device, mem_size_C);

    hipMemcpy(matrix_A_device, matrix_A_host, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(matrix_B_device, matrix_B_host, mem_size_B, hipMemcpyHostToDevice);

    // cpu_sgemm(matrix_A_host, matrix_B_host, matrix_C_host_cpu_calc, m, n, k);
    auto cpu_start = std::chrono::high_resolution_clock::now();
    cpu_sgemm(matrix_A_host, matrix_B_host, matrix_C_host_cpu_calc, m, n, k);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_duration = cpu_end - cpu_start;
    printf("CPU SGEMM time: %.3f ms\n", cpu_duration.count());


    // std::cout << &matrix_C_host_cpu_calc << std::endl;

    constexpr int BLOCK = 16;
    constexpr int STRIDE = 2;
    dim3 block(BLOCK, BLOCK);
    dim3 grid((m + BLOCK - 1) / BLOCK / STRIDE, (n + BLOCK - 1) / BLOCK / STRIDE);
    // cuda_gemm<BLOCK, BLOCK><<<grid, block>>>(matrix_A_device, matrix_B_device, matrix_C_device, m, n, k);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    cuda_gemm<BLOCK, STRIDE><<<grid, block>>>(matrix_A_device, matrix_B_device, matrix_C_device, m, n, k);
    hipEventRecord(stop);

    hipMemcpy(matrix_C_host_gpu_calc, matrix_C_device, mem_size_C, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU SGEMM time: %.9f ms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(matrix_C_host_gpu_calc, matrix_C_device, mem_size_C, hipMemcpyDeviceToHost);

    float diff = compare_matrices(m, n, matrix_C_host_gpu_calc, matrix_C_host_cpu_calc, k, n);

    printf("error: %f \n", diff);

    free(matrix_A_host);
    free(matrix_B_host);
    free(matrix_C_host_cpu_calc);
    free(matrix_C_host_gpu_calc);

    hipFree(matrix_A_device);
    hipFree(matrix_B_device);
    hipFree(matrix_C_device);

    return 0;
}